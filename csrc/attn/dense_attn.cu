#include "hip/hip_runtime.h"
// # Define TORCH_COMPILE macro

#include "kittens.cuh"
#include <hip/hip_cooperative_groups.h>
#include <iostream>
#include "../common/all.cuh"


constexpr int CONSUMER_WARPGROUPS = (3); 
constexpr int PRODUCER_WARPGROUPS = (1); 
constexpr int NUM_WARPGROUPS      = (CONSUMER_WARPGROUPS+PRODUCER_WARPGROUPS); 
constexpr int NUM_WORKERS         = (NUM_WARPGROUPS*kittens::WARPGROUP_WARPS); 

using namespace kittens;
namespace cg = cooperative_groups;

template<int D> struct fwd_attend_ker_tile_dims {};
template<> struct fwd_attend_ker_tile_dims<64> {
    constexpr static int tile_width = (64);
    constexpr static int qo_height  = (4*16);
    constexpr static int kv_height  = (8*16);
    constexpr static int stages     = (4); 
};
template<> struct fwd_attend_ker_tile_dims<128> {
    constexpr static int tile_width = (128);
    constexpr static int qo_height  = (4*16);
    constexpr static int kv_height  = (8*16);
    constexpr static int stages     = (2); 
};

template<int D> struct fwd_globals {
    using q_tile    =         st_bf<fwd_attend_ker_tile_dims<D>::qo_height, fwd_attend_ker_tile_dims<D>::tile_width>;
    using k_tile    =         st_bf<fwd_attend_ker_tile_dims<D>::kv_height, fwd_attend_ker_tile_dims<D>::tile_width>;
    using v_tile    =         st_bf<fwd_attend_ker_tile_dims<D>::kv_height, fwd_attend_ker_tile_dims<D>::tile_width>;
    using l_col_vec = col_vec<st_fl<fwd_attend_ker_tile_dims<D>::qo_height, fwd_attend_ker_tile_dims<D>::tile_width>>;
    using m_col_vec = col_vec<st_fl<fwd_attend_ker_tile_dims<D>::qo_height, fwd_attend_ker_tile_dims<D>::tile_width>>;
    using o_tile    =         st_bf<fwd_attend_ker_tile_dims<D>::qo_height, fwd_attend_ker_tile_dims<D>::tile_width>;

    using q_gl = gl<bf16,  -1, -1, -1, -1, q_tile>;
    using k_gl = gl<bf16,  -1, -1, -1, -1, k_tile>;
    using v_gl = gl<bf16,  -1, -1, -1, -1, v_tile>;
    using o_gl = gl<bf16,  -1, -1, -1, -1, o_tile>;
    using l_gl = gl<float, -1, -1, -1, -1, l_col_vec>;
    using m_gl = gl<float, -1, -1, -1, -1, m_col_vec>;

    q_gl q;
    k_gl k;
    v_gl v;
    l_gl l;
    m_gl m;
    o_gl o;

    const int kN; 
    const int hr;
};

template<int D, bool is_causal>
__global__  __launch_bounds__((NUM_WORKERS)*kittens::WARP_THREADS, 1)
void fwd_attend_ker(const __grid_constant__ fwd_globals<D> g) {
    extern __shared__ int __shm[]; 
    tma_swizzle_allocator al((int*)&__shm[0]);
    int warpid = kittens::warpid(), warpgroupid = warpid/kittens::WARPGROUP_WARPS;

    using K = fwd_attend_ker_tile_dims<D>;

    using q_tile    =         st_bf<K::qo_height, K::tile_width>;
    using k_tile    =         st_bf<K::kv_height, K::tile_width>;
    using v_tile    =         st_bf<K::kv_height, K::tile_width>;
    using l_col_vec = col_vec<st_fl<K::qo_height, K::tile_width>>;
    using m_col_vec = col_vec<st_fl<K::qo_height, K::tile_width>>;
    using o_tile    =         st_bf<K::qo_height, K::tile_width>;
    
    q_tile    (&q_smem)[CONSUMER_WARPGROUPS] = al.allocate<q_tile, CONSUMER_WARPGROUPS>();
    k_tile    (&k_smem)[K::stages]           = al.allocate<k_tile, K::stages          >();
    v_tile    (&v_smem)[K::stages]           = al.allocate<v_tile, K::stages          >();
    l_col_vec (&l_smem)[CONSUMER_WARPGROUPS] = al.allocate<l_col_vec, CONSUMER_WARPGROUPS>();
    m_col_vec (&m_smem)[CONSUMER_WARPGROUPS] = al.allocate<m_col_vec, CONSUMER_WARPGROUPS>();
    auto      (*o_smem)                      = reinterpret_cast<o_tile(*)>(q_smem);
    
    int kv_blocks   = (g.kN + K::kv_height - 1) / (K::kv_height);
    int kv_head_idx = blockIdx.y / g.hr;
    int seq_idx     = blockIdx.x * CONSUMER_WARPGROUPS; 

    __shared__ kittens::semaphore qsmem_semaphore, k_smem_arrived[K::stages], v_smem_arrived[K::stages], compute_done[K::stages];
    if (threadIdx.x == 0) { 
        init_semaphore(qsmem_semaphore, 0, 1); 
        for(int j = 0; j < K::stages; j++) {
            init_semaphore(k_smem_arrived[j], 0, 1); 
            init_semaphore(v_smem_arrived[j], 0, 1); 
            init_semaphore(compute_done[j], CONSUMER_WARPGROUPS, 0); 
        }

        tma::expect_bytes(qsmem_semaphore, sizeof(q_smem));

        for (int wg = 0; wg < CONSUMER_WARPGROUPS; wg++) {
            coord<q_tile> q_tile_idx = {blockIdx.z, blockIdx.y, (seq_idx) + wg, 0};
            tma::load_async(q_smem[wg], g.q, q_tile_idx, qsmem_semaphore);
        }

        for (int j = 0; j < K::stages - 1; j++) {
            coord<k_tile> kv_tile_idx = {blockIdx.z, kv_head_idx, j, 0};
            tma::expect_bytes(k_smem_arrived[j], sizeof(k_tile));
            tma::load_async(k_smem[j], g.k, kv_tile_idx, k_smem_arrived[j]);
            tma::expect_bytes(v_smem_arrived[j], sizeof(v_tile));
            tma::load_async(v_smem[j], g.v, kv_tile_idx, v_smem_arrived[j]);
        }
    }
    __syncthreads(); 

    int pipe_idx = K::stages - 1; 
    
    if(warpgroupid == NUM_WARPGROUPS-1) {
        warpgroup::decrease_registers<32>();      
        
        int kv_iters; 
        if constexpr (is_causal) {
            kv_iters = (seq_idx * (K::qo_height/kittens::TILE_ROW_DIM<bf16>)) - 1 + (CONSUMER_WARPGROUPS * (K::qo_height/kittens::TILE_ROW_DIM<bf16>)); 
            kv_iters = ((kv_iters / (K::kv_height/kittens::TILE_ROW_DIM<bf16>)) == 0) ? (0) : ((kv_iters / (K::kv_height/kittens::TILE_ROW_DIM<bf16>)) - 1);
        }
        else { kv_iters = kv_blocks-2; }

        if(warpid == NUM_WORKERS-4) {
            for (auto kv_idx = pipe_idx - 1; kv_idx <= kv_iters; kv_idx++) {
                coord<k_tile> kv_tile_idx = {blockIdx.z, kv_head_idx, kv_idx + 1, 0};
                tma::expect_bytes(k_smem_arrived[(kv_idx+1)%K::stages], sizeof(k_tile));
                tma::load_async(k_smem[(kv_idx+1)%K::stages], g.k, kv_tile_idx, k_smem_arrived[(kv_idx+1)%K::stages]);
                tma::expect_bytes(v_smem_arrived[(kv_idx+1)%K::stages], sizeof(v_tile));
                tma::load_async(v_smem[(kv_idx+1)%K::stages], g.v, kv_tile_idx, v_smem_arrived[(kv_idx+1)%K::stages]);
                
                kittens::wait(compute_done[(kv_idx)%K::stages], (kv_idx/K::stages)%2);
            }
        }
    }
    else {
        warpgroup::increase_registers<160>();

        rt_fl<16, K::kv_height>  att_block;
        rt_bf<16, K::kv_height>  att_block_mma;
        rt_fl<16, K::tile_width> o_reg;
        
        col_vec<rt_fl<16, K::kv_height>> max_vec, norm_vec, max_vec_last_scaled, max_vec_scaled;
        
        neg_infty(max_vec);
        zero(norm_vec);
        zero(o_reg);

        int kv_iters; 
        if constexpr (is_causal) {
            kv_iters = (seq_idx * 4) - 1 + (CONSUMER_WARPGROUPS * 4);
            kv_iters = (kv_iters/8);
        }
        else { kv_iters = kv_blocks - 1; }

        kittens::wait(qsmem_semaphore, 0);

        for (auto kv_idx = 0; kv_idx <= kv_iters; kv_idx++) {
        
            kittens::wait(k_smem_arrived[(kv_idx)%K::stages], (kv_idx/K::stages)%2);
            warpgroup::mm_ABt(att_block, q_smem[warpgroupid], k_smem[(kv_idx)%K::stages]);
            
            copy(max_vec_last_scaled, max_vec);
            if constexpr (D == 64) { mul(max_vec_last_scaled, max_vec_last_scaled, 1.44269504089f*0.125f); }
            else                   { mul(max_vec_last_scaled, max_vec_last_scaled, 1.44269504089f*0.08838834764f); }
            
            warpgroup::mma_async_wait();
            right_fill(att_block, att_block, g.k.rows - kv_idx*K::kv_height, base_types::constants<float>::neg_infty());

            if constexpr (is_causal) {
                const int q_blk = (seq_idx * (K::qo_height/kittens::TILE_ROW_DIM<bf16>)) + warpid; 
                      int k_blk = (kv_idx * (K::kv_height/kittens::TILE_ROW_DIM<bf16>)); 

                #pragma unroll
                for(int _ = 0; k_blk == (kv_iters-1)*(K::kv_height/kittens::TILE_ROW_DIM<bf16>) || k_blk == (kv_iters)*(K::kv_height/kittens::TILE_ROW_DIM<bf16>); k_blk+=10000) {
                    #pragma unroll
                    for (auto j = 0; j < (K::kv_height/kittens::TILE_ROW_DIM<bf16>); j++) {
                        auto k_idx = k_blk + j;
                        auto &attn_subtile = reinterpret_cast<rt_fl<16, 16>&>(att_block.tiles[0][j]);

                        if      (k_idx >  q_blk) { neg_infty  (attn_subtile); }
                        else if (k_idx == q_blk) { make_causal(attn_subtile, attn_subtile, kittens::base_types::constants<float>::neg_infty()); }
                        __syncwarp();
                    }
                }
            }

            row_max(max_vec, att_block, max_vec);
            
            if constexpr (D == 64) { 
                mul(att_block, att_block,    1.44269504089f*0.125f); 
                mul(max_vec_scaled, max_vec, 1.44269504089f*0.125f);
            }
            else                   { 
                mul(att_block, att_block,    1.44269504089f*0.08838834764f); 
                mul(max_vec_scaled, max_vec, 1.44269504089f*0.08838834764f);
            }

            sub_row(att_block, att_block, max_vec_scaled);
            exp2(att_block, att_block);
            sub(max_vec_last_scaled, max_vec_last_scaled, max_vec_scaled);
            exp2(max_vec_last_scaled,       max_vec_last_scaled);
            mul(norm_vec,            norm_vec,     max_vec_last_scaled);
            row_sum(norm_vec,  att_block, norm_vec);
            // add(att_block, att_block, 0.f);
            copy(att_block_mma, att_block); 
            mul_row(o_reg, o_reg, max_vec_last_scaled); 

            kittens::wait(v_smem_arrived[(kv_idx)%K::stages], (kv_idx/K::stages)%2); 

            warpgroup::mma_AB(o_reg, att_block_mma, v_smem[(kv_idx)%K::stages]);
            warpgroup::mma_async_wait();

            if(warpgroup::laneid() == 0) arrive(compute_done[(kv_idx)%K::stages], 1);
        }

        div_row(o_reg, o_reg, norm_vec);
        warpgroup::store(o_smem[warpgroupid], o_reg); 
        warpgroup::sync(warpgroupid+4);

        if (warpid % 4 == 0) {
            coord<o_tile> o_tile_idx = {blockIdx.z, blockIdx.y, (seq_idx) + warpgroupid, 0};
            tma::store_async(g.o, o_smem[warpgroupid], o_tile_idx);
        }

        // store as single constant
        exp2(max_vec_scaled, max_vec_scaled);
        mul(max_vec_scaled, max_vec_scaled, norm_vec);
        unary_op<chipmunk::base_ops::rcp>(max_vec_scaled, max_vec_scaled);
        warpgroup::store(l_smem[warpgroupid], max_vec_scaled);
        warpgroup::sync(warpgroupid+4);

        if (warpid % 4 == 0) {
            coord<l_col_vec> tile_idx = {blockIdx.z, blockIdx.y, 0, (seq_idx) + warpgroupid};
            tma::store_async(g.l, l_smem[warpgroupid], tile_idx);
        }
        tma::store_async_wait();
    }
}

#ifdef TORCH_COMPILE

#include "pyutils/torch_helpers.cuh"
#include <ATen/cuda/HIPContext.h>
#include <iostream>

namespace chipmunk {
void
dense_attn(at::Tensor q, at::Tensor k, at::Tensor v, at::Tensor o, at::Tensor l_vec)
{
    // CHECK_INPUT(q);
    // CHECK_INPUT(k);
    // CHECK_INPUT(v);

    auto batch    = q.size(0);
    auto seq_len  = q.size(2); 
    auto kseq_len = k.size(2);
    auto head_dim = q.size(3); 
    auto is_causal = false; 
    auto qo_heads = q.size(1);
    auto kv_heads = k.size(1);

    // check to see that these dimensions match for all inputs
    TORCH_CHECK(q.size(0) == batch, "Q batch dimension - idx 0 - must match for all inputs");
    TORCH_CHECK(k.size(0) == batch, "K batch dimension - idx 0 - must match for all inputs");
    TORCH_CHECK(v.size(0) == batch, "V batch dimension - idx 0 - must match for all inputs");

    TORCH_CHECK(q.size(2) == seq_len, "Q sequence length dimension - idx 2 - must match for all inputs");
    TORCH_CHECK(k.size(2) == kseq_len, "K sequence length dimension - idx 2 - must match for all inputs");
    TORCH_CHECK(v.size(2) == kseq_len, "V sequence length dimension - idx 2 - must match for all inputs");

    TORCH_CHECK(q.size(3) == head_dim, "Q head dimension - idx 3 - must match for all non-vector inputs");
    TORCH_CHECK(k.size(3) == head_dim, "K head dimension - idx 3 - must match for all non-vector inputs");
    TORCH_CHECK(v.size(3) == head_dim, "V head dimension - idx 3 - must match for all non-vector inputs");

    TORCH_CHECK(qo_heads >= kv_heads, "QO heads must be greater than or equal to KV heads");
    TORCH_CHECK(qo_heads % kv_heads == 0, "QO heads must be divisible by KV heads");
    TORCH_CHECK(q.size(1) == qo_heads, "QO head dimension - idx 1 - must match for all inputs");
    TORCH_CHECK(k.size(1) == kv_heads, "KV head dimension - idx 1 - must match for all inputs");
    TORCH_CHECK(v.size(1) == kv_heads, "KV head dimension - idx 1 - must match for all inputs");  

    auto hr = qo_heads / kv_heads;

    c10::BFloat16* q_ptr = q.data_ptr<c10::BFloat16>();
    c10::BFloat16* k_ptr = k.data_ptr<c10::BFloat16>();
    c10::BFloat16* v_ptr = v.data_ptr<c10::BFloat16>();

    bf16*  d_q = reinterpret_cast<bf16*>(q_ptr);
    bf16*  d_k = reinterpret_cast<bf16*>(k_ptr);
    bf16*  d_v = reinterpret_cast<bf16*>(v_ptr);
    
    // for the returned outputs
    // at::Tensor o     = torch::empty({static_cast<const uint>(batch), 
    //                                     static_cast<const uint>(qo_heads), 
    //                                     static_cast<const uint>(seq_len), 
    //                                     static_cast<const uint>(head_dim)}, v.options().memory_format(at::MemoryFormat::Contiguous));
    
    // at::Tensor l_vec = torch::empty({static_cast<const uint>(batch), 
    //                                     static_cast<const uint>(qo_heads), 
    //                                     static_cast<const uint>(seq_len), 
    //                                     static_cast<const uint>(1)}, 
    //                                     torch::TensorOptions().dtype(torch::kFloat).device(q.device()).memory_format(at::MemoryFormat::Contiguous));
    at::Tensor m_vec = torch::empty({static_cast<const uint>(batch), 
                                        static_cast<const uint>(qo_heads), 
                                        static_cast<const uint>(seq_len), 
                                        static_cast<const uint>(1)}, 
                                        torch::TensorOptions().dtype(torch::kFloat).device(q.device()).memory_format(at::MemoryFormat::Contiguous));
        

    bf16*  o_ptr = reinterpret_cast<bf16*>(o.data_ptr<c10::BFloat16>());
    bf16*  d_o   = reinterpret_cast<bf16*>(o_ptr);

    float* l_ptr = reinterpret_cast<float*>(l_vec.data_ptr<float>());
    float* d_l   = reinterpret_cast<float*>(l_ptr);

    float* m_ptr = reinterpret_cast<float*>(m_vec.data_ptr<float>());
    float* d_m   = reinterpret_cast<float*>(m_ptr);

    auto stream = at::cuda::getCurrentCUDAStream().stream(); 

    if (head_dim != 128) {
        throw std::runtime_error("Head dimension must be 128");
    }
    using q_tile    =         st_bf<fwd_attend_ker_tile_dims<128>::qo_height, fwd_attend_ker_tile_dims<128>::tile_width>;
    using k_tile    =         st_bf<fwd_attend_ker_tile_dims<128>::kv_height, fwd_attend_ker_tile_dims<128>::tile_width>;
    using v_tile    =         st_bf<fwd_attend_ker_tile_dims<128>::kv_height, fwd_attend_ker_tile_dims<128>::tile_width>;
    using l_col_vec = col_vec<st_fl<fwd_attend_ker_tile_dims<128>::qo_height, fwd_attend_ker_tile_dims<128>::tile_width>>;
    using o_tile    =         st_bf<fwd_attend_ker_tile_dims<128>::qo_height, fwd_attend_ker_tile_dims<128>::tile_width>;

    using q_global = gl<bf16,  -1, -1, -1, -1, q_tile>;
    using k_global = gl<bf16,  -1, -1, -1, -1, k_tile>;
    using v_global = gl<bf16,  -1, -1, -1, -1, v_tile>;
    using l_global = gl<float, -1, -1, -1, -1, l_col_vec>;
    using o_global = gl<bf16,  -1, -1, -1, -1, o_tile>;

    using globals      = fwd_globals<128>;

    if (is_causal) {
        throw std::runtime_error("Causal attention is not supported yet.");
    }


    q_global qg_arg{d_q, static_cast<unsigned int>(batch), static_cast<unsigned int>(qo_heads), static_cast<unsigned int>(seq_len), 128U};
    k_global kg_arg{d_k, static_cast<unsigned int>(batch), static_cast<unsigned int>(kv_heads), static_cast<unsigned int>(kseq_len), 128U};
    v_global vg_arg{d_v, static_cast<unsigned int>(batch), static_cast<unsigned int>(kv_heads), static_cast<unsigned int>(kseq_len), 128U};
    l_global lg_arg{d_l, static_cast<unsigned int>(batch), static_cast<unsigned int>(qo_heads), 1U,   static_cast<unsigned int>(seq_len)};
    l_global mg_arg{d_m, static_cast<unsigned int>(batch), static_cast<unsigned int>(qo_heads), 1U,   static_cast<unsigned int>(seq_len)};
    o_global og_arg{d_o, static_cast<unsigned int>(batch), static_cast<unsigned int>(qo_heads), static_cast<unsigned int>(seq_len), 128U};
    chipmunk::create_tensor_map_with_strides<q_tile, 2>(&qg_arg.tma_descs.tma_desc, d_q, batch, qo_heads, seq_len, head_dim, q.stride(0), q.stride(1), q.stride(2));
    chipmunk::create_tensor_map_with_strides<k_tile, 2>(&kg_arg.tma_descs.tma_desc, d_k, batch, kv_heads, kseq_len, head_dim, k.stride(0), k.stride(1), k.stride(2));
    chipmunk::create_tensor_map_with_strides<v_tile, 2>(&vg_arg.tma_descs.tma_desc, d_v, batch, kv_heads, kseq_len, head_dim, v.stride(0), v.stride(1), v.stride(2));
    chipmunk::create_tensor_map_with_strides<o_tile, 2>(&og_arg.tma_descs.tma_desc, d_o, batch, qo_heads, seq_len, head_dim, o.stride(0), o.stride(1), o.stride(2));
    globals g{qg_arg, kg_arg, vg_arg, lg_arg, mg_arg, og_arg, static_cast<int>(kseq_len), static_cast<int>(hr)};

    auto mem_size = kittens::MAX_SHARED_MEMORY;
    auto threads  = NUM_WORKERS * kittens::WARP_THREADS;

    // TORCH_CHECK(seq_len % (CONSUMER_WARPGROUPS*kittens::TILE_DIM*4) == 0, "sequence length must be divisible by 192");
    auto num_tokens_per_block = CONSUMER_WARPGROUPS*kittens::TILE_ROW_DIM<bf16>*4;
    dim3 grid((seq_len+num_tokens_per_block-1)/num_tokens_per_block, qo_heads, batch);

    auto ker_template = fwd_attend_ker<128, false>;

    hipFuncSetAttribute(reinterpret_cast<const void*>(
        ker_template),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        mem_size
    );

    ker_template<<<grid, (32*NUM_WORKERS), mem_size, stream>>>(g);

    CHECK_CUDA_ERROR(hipGetLastError());
}
}

#else

#include "harness.impl"

#endif